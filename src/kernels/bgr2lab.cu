#include <hip/hip_runtime.h>

/**
 * TODO: Document operation
 * TODO: Look for potential thrust implementation of this function
 */
__global__
void gpuBgrToLab(
        unsigned char* devBgr, 
        unsigned char* devL, unsigned char* devA, unsigned char* devB, 
        int rows, int cols, float gamma
        )
{
    // Compute pixel row, column indices
    const int col = blockDim.x * blockIdx.x + threadIdx.x;
    const int row = blockDim.y * blockIdx.y + threadIdx.y;


    if( row >= rows || col >= cols )
    {
        return;
    }

    // Threshold for RBG2Lab transform delta = (6 / 29) ** 3
    const float DELTA = 0.008856;
    // slope value is 1/(3 delta ** 2) and the intercept is 4/29
    const float DELTA_SLOPE = 7.787037;
    const float DELTA_INTERCEPT = 4.0 / 29.0;

    // Constants for LAB space
    const float L_MAX = 100.0;
    const float AB_MIN = -75.0;
    const float AB_MAX = 93.0;

    // Pixel index
    int c = row * cols + col;
    int idx = 3 * c;

    // Normalize to 0, 1 and apply gamma corrections
    float blue = (devBgr[idx]) / 255.0;
    float green = (devBgr[idx + 1]) / 255.0;
    float red = (devBgr[idx + 2]) / 255.0;

    blue = powf(blue, gamma);
    green = powf(green, gamma);
    red = powf(red, gamma);

    float x = (0.412453 * red) +  (0.357580 * green) + (0.180423 * blue);
    float y = (0.212671 * red) +  (0.715160 * green) + (0.072169 * blue);
    float z = (0.019334 * red) +  (0.119193 * green) + (0.950227 * blue);

    // Set white point = D65 (0.950456, 1.000000, 1.088754), 
    // y is not divided since it's coefficient is 1.000000
    // Read:
    // [1] https://en.wikipedia.org/wiki/Illuminant_D65
    // [2] https://en.wikipedia.org/wiki/White_point
    // [3] https://en.wikipedia.org/wiki/Lab_color_space#Forward_transformation
    x /= 0.950456;
    z /= 1.088754;
    
    // cbrt is the fast cube-root routine
    float fx = (x > DELTA) ? cbrtf(x) : (DELTA_SLOPE * x + DELTA_INTERCEPT);
    float fy = (y > DELTA) ? cbrtf(y) : (DELTA_SLOPE * y + DELTA_INTERCEPT);
    float fz = (z > DELTA) ? cbrtf(z) : (DELTA_SLOPE * z + DELTA_INTERCEPT);

    // CIEXYZ -> CIEL*a*b* transformation
    float l = (y > DELTA) ? (116 * cbrtf(y) - 16.0) : (903.3 * y);
    float a = 500.0 * (fx - fy);
    float b = 200.0 * (fy - fz);

    // Normalize L*, a*, b* values so they all lie in [0, 1]
    l /= L_MAX;
    a = (a - AB_MIN) / (AB_MAX - AB_MIN);
    b = (b - AB_MIN) / (AB_MAX - AB_MIN);
    l = (l < 0) ? 0 : ( l > 1 ? 1 : l);
    a = (a < 0) ? 0 : ( a > 1 ? 1 : a);
    b = (b < 0) ? 0 : ( b > 1 ? 1 : b);
    
    devL[c] = (unsigned char) (255 * l);
    devA[c] = (unsigned char) (255 * a);
    devB[c] = (unsigned char) (255 * b);
}


